#include "hip/hip_runtime.h"
#include "lab1.h"


struct Lab1VideoGenerator::Impl {
    int t = 0;
};

void Lab1VideoGenerator::generateNoise(float * noiseArr, float freq) {
    int noise_width = W * 2;
    int noise_height = H * 2;

    // Generate a noise value for each pixel
    float invWidth = 1.0f / float(noise_width);
    float invHeight = 1.0f / float(noise_height);
    float noise;
    float min = 0.0f;
    float max = 0.0f;

    for (int x=0; x<noise_width; ++x) for (int y=0; y<noise_height; ++y) {

        noise = noiseMaker->getFractal(float(x)*invWidth, float(y)*invHeight, freq);

        noiseArr[y*noise_width + x] = noise;

        // Keep track of minimum and maximum noise values
        if (noise < min) min = noise;
        if (noise > max) max = noise;
    }

    // Convert noise values to pixel colour values.
    float temp = 1.0f / (max - min);

    for (int x=0; x<noise_width; ++x) for (int y=0; y<noise_height; ++y) {

        // "Stretch" the gaussian distribution of noise values to better fill -1 to 1 range.
        noise = noiseArr[y*noise_width + x];
        noise = -1.0f + 2.0f*(noise - min)*temp;
        // Remap to RGB friendly colour values in range between 0 and 1.
        noise += 1.0f;
        noise *= 0.5f;
        noiseArr[y*noise_width + x] = noise;
    }	
}


float Lab1VideoGenerator::getNoise(float * noiseArr, int x, int y) {
    int noise_width = W * 2;
    return noiseArr[(y + H /2 ) * noise_width + (x + W / 2 )];
}

void Lab1VideoGenerator::setRotMatrix(int degree) {
    rotMat[0][0] = cos(degree * M_PI / 180);
    rotMat[0][1] = -sin(degree * M_PI / 180);
    rotMat[1][0] = sin(degree * M_PI / 180);
    rotMat[1][1] = cos(degree * M_PI / 180);
}

void Lab1VideoGenerator::rotate(int &x, int &y) {
    // normalize it
    float nx = float(x) / W - 0.5;
    float ny = float(y) / H - 0.5;
    x = int((nx * rotMat[0][0] + ny * rotMat[0][1] + 0.5) * W);
    y = int((nx * rotMat[1][0] + ny * rotMat[1][1] + 0.5) * H);
}

Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
    noiseMaker = new Perlin2D();
    loose_noise = new float[W*2*H*2];
    dense_noise = new float[W*2*H*2];
    generateNoise(loose_noise, 1.0);
    generateNoise(dense_noise, 8.0);
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
    info.w = W;
    info.h = H;
    info.n_frame = NFRAME;
    // fps = 24/1 = 24
    info.fps_n = fps;
    info.fps_d = 1;
};

void Lab1VideoGenerator::Generate(uint8_t *yuv) {
    int loop = fps * 2;
    float w = float(impl->t % loop) / loop;
    w = w * w;
    int direction = impl->t / loop % 2;
    setRotMatrix(impl->t * 24 / fps);
    for(int i=0 ; i<W*H ; i++) {
        int x = i % W;
        int y = i / W;
        rotate(x, y); 
        float n1 = getNoise(loose_noise, x, y);
        float n2 = getNoise(dense_noise, x, y);
        float color;
        if(direction == 0)
            color = (1.0 - w) * n1 + w * n2;
        else 
            color = w * n1 + (1.0 - w) * n2;
        hipMemset(yuv+i, color * 255, 1);
    }
    hipMemset(yuv+W*H, 128, W*H/2);
    impl->t++;
}

